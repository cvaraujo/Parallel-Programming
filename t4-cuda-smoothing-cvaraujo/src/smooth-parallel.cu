
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define MASK_WIDTH 15
#define TILE_WIDTH 16

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

void check_cuda(hipError_t error, const char *filename, const int line)
{
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line,
                 hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

void writePPM(PPMImage *img) {

  fprintf(stdout, "P6\n");
  fprintf(stdout, "# %s\n", COMMENT);
  fprintf(stdout, "%d %d\n", img->x, img->y);
  fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

  fwrite(img->data, 3 * img->x, img->y, stdout);
  fclose(stdout);
}


__global__ void smoothing_kernel(PPMPixel *image, PPMPixel *image_copy, int rows, int cols) {
    long long int col = blockIdx.x * blockDim.x + threadIdx.x;
    long long int row = blockIdx.y * blockDim.y + threadIdx.y;

    int border = (int)((MASK_WIDTH - 1) / 2);
    const int dimension = TILE_WIDTH + (MASK_WIDTH - 1);

    __shared__ PPMPixel shared[dimension+1][dimension+1];

    int size = ceil((float)(dimension * dimension) / (blockDim.x * blockDim.y));

    int block_start_col = blockIdx.x * blockDim.x;
    int block_start_row = blockIdx.y * blockDim.y;

    for(int k = 0; k < size; k++){

    	int index = (threadIdx.y * blockDim.x) + threadIdx.x + (k * blockDim.x * blockDim.y);

        int x = (int)(index / dimension);
        int y = (index % dimension);

        if(x < dimension && y < dimension){
            int sr = block_start_row + x - border;
            int sc = block_start_col + y - border;

            if(sr >= 0 && sc >= 0 && sc < cols && sr < rows) shared[x][y] = image_copy[sr * cols + sc];
            else shared[x][y].red = shared[x][y].green = shared[x][y].blue = 0;
            
        }
    }

    __syncthreads();

    int total_red, total_blue, total_green;
    total_red = total_blue = total_green = 0;
    //Determiando a posicao da matriz
    if(row < rows && col < cols){
        for(int i = threadIdx.y; i < (threadIdx.y + MASK_WIDTH); i++){
            for(int j = threadIdx.x; j < (threadIdx.x + MASK_WIDTH); j++){
                total_red += shared[i][j].red;
                total_green += shared[i][j].green;
                total_blue += shared[i][j].blue;
            } 
        } 

        image[row * cols + col].red = total_red / (MASK_WIDTH*MASK_WIDTH);
        image[row * cols + col].blue = total_blue / (MASK_WIDTH*MASK_WIDTH);
        image[row * cols + col].green = total_green / (MASK_WIDTH*MASK_WIDTH);

    }
}

void Smoothing(PPMImage *image, PPMImage *image_copy) {
  hipEvent_t start, stop;
  PPMPixel *_image;
  PPMPixel *_image_copy;

  hipMalloc(&_image, sizeof(PPMPixel) * image->x * image->y);
  hipMalloc(&_image_copy, sizeof(PPMPixel) * image->x * image->y);

  hipMemcpy(_image, image->data, image->x * image->y * sizeof(PPMPixel), hipMemcpyHostToDevice);

  // Create Events
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  dim3 dimGrid((image->x + (TILE_WIDTH-1))/TILE_WIDTH, (image->y + (TILE_WIDTH-1))/TILE_WIDTH, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

  CUDACHECK(hipEventRecord(start));
  smoothing_kernel<<<dimGrid, dimBlock>>>(_image_copy, _image, image->y, image->x);
  
  CUDACHECK(hipEventRecord(stop));
  CUDACHECK(hipEventSynchronize(stop));

  hipMemcpy(image->data, _image_copy, sizeof(PPMPixel) * image->y * image->x, hipMemcpyDeviceToHost);

  // Destroy events
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));
}

int main(int argc, char *argv[]) {
  FILE *input;
  char filename[255];
  double t;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open input file!\n");
    return 1;
  }

  // Read input filename
  fscanf(input, "%s\n", filename);

  // Read input file
  PPMImage *image = readPPM(filename);
  PPMImage *image_output = readPPM(filename);

  // Call Smoothing Kernel
  t = omp_get_wtime();
  Smoothing(image_output, image);
  t = omp_get_wtime() - t;

  // Write result to stdout
  writePPM(image_output);

  // Print time to stderr
  fprintf(stderr, "%lf\n", t);

  // Cleanup
  free(image);
  free(image_output);

  return 0;
}
