
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

const int THR = 1024;

__global__ void matrix_sum(int *C, int *A, int *B, int n) {
	   int index = blockIdx.x * blockDim.x + threadIdx.x;
	   if (index < n) C[index] = A[index] + B[index];	   
}

int main(int argc, char **argv) {
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  int i, j;
  double t;

  // Input
  int rows, cols;
  FILE *input;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return EXIT_FAILURE;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open file\n");
    return EXIT_FAILURE;
  }

  fscanf(input, "%d", &rows);
  fscanf(input, "%d", &cols);

  // Allocate memory on the host
  h_A = (int *)malloc(sizeof(int) * rows * cols); //host_init(h_A);
  h_B = (int *)malloc(sizeof(int) * rows * cols); //host_init(h_B);
  h_C = (int *)malloc(sizeof(int) * rows * cols); //host_init(h_C);

  // Initialize memory
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      h_A[i * cols + j] = h_B[i * cols + j] = i + j;
    }
  }

  int bytes = sizeof(int) * rows * cols;
  // Copy data to device
  // ...
  hipMalloc(&d_A, bytes);
  hipMalloc(&d_B, bytes);
  hipMalloc(&d_C, bytes);

  hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

  // Compute matrix sum on device
  // Leave only the kernel and synchronize inside the timing region!
  int N = rows * cols;

  t = omp_get_wtime();
  matrix_sum<<<(N + THR-1)/THR, THR>>>(d_C, d_A, d_B, rows * cols);	
  hipDeviceSynchronize();
  t = omp_get_wtime() - t;
  

  // Copy data back to host
  // ...
  hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

  long long int sum = 0;

  // Keep this computation on the CPU
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      sum += h_C[i * cols + j];
    }
  }

  fprintf(stdout, "%lli\n", sum);
  fprintf(stderr, "%lf\n", t);

  hipFree(d_A); free(h_A);
  hipFree(d_B); free(h_B);
  hipFree(d_C); free(h_C);

  return 0;
}
