
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

#define BLOCK_SIZE 32

void check_cuda(hipError_t error, const char *filename, const int line)
{
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line,
                 hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

__global__ void histogram_kernel(PPMPixel* data, int rows, int cols, float* h) {
  // Local variables
  long long tid = threadIdx.x + blockDim.x * blockIdx.x;  
  int stride = blockDim.x * gridDim.x;
  int i = threadIdx.x;
  long long n = rows * cols;

  __shared__ int temp[64];
  temp[i%64] = 0;
  
  __syncthreads();

  while (tid < n) {
    atomicAdd(&(temp[int(data[tid].red * 16 + data[tid].green * 4  + data[tid].blue)]), 1);
    tid += stride;
  }
  __syncthreads();
  
  atomicAdd(&(h[i%64]), temp[i%64]);

}

double Histogram(PPMImage *image, float *h_h) {
  float ms;
  hipEvent_t start, stop;
  PPMPixel *d_pixels;
  float *h_d;
  long long  n = image->y * image->x;
  int i;

  for (i = 0; i < n; i++) {
    image->data[i].red = floor((image->data[i].red * 4) / 256);
    image->data[i].green = floor((image->data[i].green * 4) / 256);
    image->data[i].blue = floor((image->data[i].blue * 4) / 256);
  }

  hipMalloc(&d_pixels, sizeof(PPMPixel) * n);
  hipMalloc(&h_d, sizeof(float)*64);

  hipMemcpy(d_pixels, image->data, n * sizeof(PPMPixel), hipMemcpyHostToDevice);
  hipMemcpy(h_d, h_h, 64 * sizeof(float), hipMemcpyHostToDevice);

  // Create Events
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));
  
  int THR = 64;
  int BLO = (n + (THR-1))/THR;
  
  // Launch kernel and compute kernel runtime.
  // Warning: make sure only the kernel is being profiled, memcpies should be
  // out of this region.
  size_t t_size = 64 * sizeof(float);

  CUDACHECK(hipEventRecord(start));
  histogram_kernel<<<BLO, THR,t_size>>>(d_pixels, image->x, image->y, h_d);
  CUDACHECK(hipEventRecord(stop));
  CUDACHECK(hipEventSynchronize(stop));
  CUDACHECK(hipEventElapsedTime(&ms, start, stop));

  hipMemcpy(h_h, h_d, 64 * sizeof(float), hipMemcpyDeviceToHost);

  // Destroy events
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));
  hipFree(h_d);

  return ((double)ms) / 1000.0;
}

int main(int argc, char *argv[]) {

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  PPMImage *image = readPPM(argv[1]);
  float *h = (float *)malloc(sizeof(float) * 64);
  long long  n = image->y * image->x;

  // Initialize histogram
  for (int i = 0; i < 64; i++)
    h[i] = 0.0;
  
  // Compute histogram
  double t = Histogram(image, h);

  for (int i = 0; i < 64; i++)
    printf("%0.3f ", h[i]/n);
  printf("\n");

  fprintf(stderr, "%lf\n", t);
  free(h);
}
